
#include <hip/hip_runtime.h>
extern "C"{
__global__ void row_sum(const double* A, double* rowSums, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        double sumExp = 0.0;
        for (int col = 0; col < N; ++col) {
            sumExp += exp(A[row * N + col]);
        }
        rowSums[row] = sumExp;
    }
}
}