#include "hip/hip_runtime.h"
#include "test.h"

void compute_ground_truth(
    double* hA, 
    double* rowSums,
    double* hOut_gt, 
    int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            hOut_gt[i*N+j] = exp(hA[i*N+j]) / rowSums[i];
        }
    }
}

bool Softmax::_run(
    hipFunction_t kernel
){
    hipDeviceptr_t dA, dB, dOut, dOut_gt;
    int M = 128, N = 128;
    size_t i;
    double * hA, * hB, * hOut, * hOut_gt;
    
    hA = (double*)malloc(M * N * sizeof(double));
    hB = (double*)malloc(M * sizeof(double));
    hOut = (double*)malloc(M * N * sizeof(double));
    hOut_gt = (double*)malloc(M * N * sizeof(double));

    void* args[5];

    // Generate input for execution, and create output buffers.
    for (i = 0; i < M * N; ++i) {
        hA[i] = (double)(1 - 2 * ((i & 1) << 1));
    }
    for (i = 0; i < M; ++i) {
        hB[i] = (double)i + 1;
    }
    CUDA_SAFE_CALL(hipMalloc(&dA, M * N * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dB, M * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc(&dOut, M * N * sizeof(double)));    
    
    CUDA_SAFE_CALL(hipMemcpyHtoD(dA, hA, M * N * sizeof(double)));
    CUDA_SAFE_CALL(hipMemcpyHtoD(dB, hB, M * sizeof(double)));

    args[0] = &dA;
    args[1] = &dB;
    args[2] = &dOut;
    args[3] = &M;
    args[4] = &N;

    CUDA_SAFE_CALL( hipModuleLaunchKernel(kernel,
                                (M*N + 31) / 32,  1, 1, // grid dim
                                32, 1, 1, // block dim
                                0, NULL, // shared mem and stream
                                args, 0)); // arguments
    CUDA_SAFE_CALL(hipCtxSynchronize()); // Retrieve and print output.
    CUDA_SAFE_CALL(hipMemcpyDtoH(hOut, dOut, M * N * sizeof(double)));

    compute_ground_truth(hA, hB, hOut_gt, M, N);
    bool is_all_close = all_close(hOut, hOut_gt, M * N);
    
    // Release resources.
    CUDA_SAFE_CALL(hipFree(dA));
    CUDA_SAFE_CALL(hipFree(dB));
    CUDA_SAFE_CALL(hipFree(dOut));
    free(hA);
    free(hB);
    free(hOut);
    free(hOut_gt);
    return is_all_close;

}
